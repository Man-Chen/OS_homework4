#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

// Define constants and data types

//1MB in global memory
#define STORAGE_SIZE        1085440

// file's maximum size 
#define MAX_FILE_SIZE       1048576

// Data File I/O
#define OUTPUTFILE          "./snapshot.bin"
#define DATAFILE            "./data.bin"


typedef unsigned char uchar;
typedef uint32_t u32;

// G_WRITE mode is 1
const int G_WRITE   = 1;
// G_READ mode is 2
const int G_READ    = 2;

// list all file by size order
const int LS_S      = 3;
// list all file by modified time order
const int LS_D      = 4;
const int RM        = 5;
const int RM_RF     = 6;

// Declare variables
__device__ __managed__ uchar *volume;

// Initialize Function
// ******************************************************************
void init_volume() {
    memset(volume, 0, STORAGE_SIZE*sizeof(uchar));
}

// ******************************************************************

// ******************************************************************
// File I/O Function

// File output use stdio.h Function fopen,fwrite,fclose
void writeBinaryFile(char *fileName, uchar *input, int fileSize) {
    FILE *fptr = fopen(fileName, "wb");
    // Read data from input file
    fwrite(input, sizeof(unsigned char), fileSize, fptr);
	fclose(fptr);
}

// File input use stdio.h Function fopen,fseek,ftell,rewind,fread,fclose
int loadBinaryFile(char *fileName, uchar *input, int fileSize) {
    FILE *fptr = fopen(fileName, "rb");
    // Get size
    fseek(fptr, 0, SEEK_END);
    int size = ftell(fptr);
    rewind(fptr);
    // Read data from input file
    fread(input, sizeof(unsigned char), size, fptr);
    if (fileSize < size) {
        printf("ERROR: Input size is illegal!\n");
    }
	fclose(fptr);
    return size;
}

// ******************************************************************

// ******************************************************************
// File System Operation
__device__ u32 open(char *name, int type) {
    u32 fp = 0;
    printf("Open %s %d\n", name, type);
    return fp;
}

__device__ void write(uchar *src, int len, u32 fp) {
    // Not implement
    printf("Write %s %d %d\n", src, len, fp);
}

__device__ void read(uchar *dst, int len, u32 fp) {
    // Not implement
    printf("Read %s %d %d\n", dst, len, fp);
}


// ******************************************************************

// ******************************************************************
// Kernel function
__global__ void mykernel(uchar *input, uchar *output) {
    //####kernel start####
    u32 fp = open("t.txt\0", G_WRITE);
    write(input, 64, fp);
    fp = open("b.txt\0", G_WRITE);
    write(input+32, 32, fp);
    fp = open("t.txt\0", G_WRITE);
    write(input+32, 32, fp);
    read(output, 32, fp);
    fp = open("b.txt\0", G_WRITE);
    write(input+64, 12, fp);
	gsys(LS_S);
    gsys(LS_D);
    gsys(RM, "t.txt\0");
    gsys(LS_S);
	
	//Bonus Test
	/*fp = open("a.txt\0",G_WRITE);
	write(input+128, 64, fp);
	fp = open("b.txt\0",G_WRITE);
	write(input+256, 32, fp);
	gsys(MKDIR, "soft\0");
	gsys(LS_S);
    gsys(LS_D);
	gsys(CD, "soft\0");
	gsys(PWD);
	*/
    //####kernel end####
}
// ******************************************************************

int main() {
    hipMallocManaged(&volume, STORAGE_SIZE);
    init_volume();

    uchar *input, *output;
    hipMallocManaged(&input, MAX_FILE_SIZE);
    hipMallocManaged(&output, MAX_FILE_SIZE);
    for (int i = 0; i < MAX_FILE_SIZE; i++) {
        output[i] = 0;
    }
    loadBinaryFile(DATAFILE, input, MAX_FILE_SIZE);

    hipSetDevice(4);
    mykernel<<<1, 1>>>(input, output);
    hipDeviceSynchronize();
    writeBinaryFile(OUTPUTFILE, output, MAX_FILE_SIZE);
    hipDeviceReset();

    return 0;
}
